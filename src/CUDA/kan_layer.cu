#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "../KAN/kan_layer.h"



__device__ double cuda_b_spline_basis(unsigned int i, unsigned int k, double x, const double* knots) {
    if (k == 0) {
        return knots[i] <= x && x < knots[i + 1] ? 1.0 : 0.0;
    }
    double coef1 = (x - knots[i]) / (knots[i + k] - knots[i]);
    double coef2 = (knots[i + k + 1] - x) / (knots[i + k + 1] - knots[i + 1]);
    return coef1 * cuda_b_spline_basis(i, k - 1, x, knots) + coef2 * cuda_b_spline_basis(i + 1, k - 1, x, knots);
}

__global__ void calcSplineValuesKernel(double* x, double* grid, unsigned int grid_size, unsigned int spline_order, double* B, unsigned int batch_size, unsigned int in_size, double* knots) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx < in_size && idy < batch_size) {
        for (int k = 0; k < grid_size + spline_order; k++) {
            unsigned int n = grid_size + spline_order - 1;

            double result = 0.0;
            for (int i = 0; i < n; i++) {
                result += grid[idx * (grid_size + 2 * spline_order + 1) + i] * cuda_b_spline_basis(i, spline_order, x[idy * in_size + idx], knots);
            }

            B[(idx * batch_size + idy) * (grid_size + spline_order) + k] = result;
        }
    }
}

__global__ void fitCoefficientsKernel(double* B, double* y, double* coefs, unsigned int batch_size, unsigned int in_size, unsigned int grid_size, unsigned int spline_order, unsigned int out_size, double l2_reg) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < in_size) {
        for (int j = 0; j < out_size; j++) {
            for (int k = 0; k < grid_size + spline_order; k++) {
                double sum = 0.0;
                for (int b = 0; b < batch_size; b++) {
                    sum += B[(idx * batch_size + b) * (grid_size + spline_order) + k] * y[(idx * batch_size + b) * out_size + j];
                }
                coefs[(idx * (grid_size + spline_order) + k) * out_size + j] = sum / (batch_size + l2_reg);
            }
        }
    }
}


extern "C" void fitSplineCoefficients(KANDenseLayer* layer, double* x, double* y, double* grid, unsigned int batch_size, double l2_reg);
void fitSplineCoefficients(KANDenseLayer* layer, double* x, double* y, double* grid, unsigned int batch_size, double l2_reg) {
    unsigned int grid_size = layer->grid_size;
    unsigned int spline_order = layer->spline_order;
    unsigned int in_size = layer->in;
    unsigned int out_size = layer->out;
    unsigned int n = grid_size + spline_order - 1;

    double* h_knots = (double*)calloc(n + spline_order + 1, sizeof(double));
    for (unsigned int i = 0; i < n + spline_order + 1; i++) {
        h_knots[i] = (double)i / (n + spline_order);
    }

    double* d_x;
    double* d_y;
    double* d_grid;
    double* d_B;
    double* d_coefs;
    double* d_knots;

    hipMalloc(&d_x, batch_size * in_size * sizeof(double));
    hipMalloc(&d_y, batch_size * in_size * out_size * sizeof(double));
    hipMalloc(&d_grid, in_size * (grid_size + 2 * spline_order + 1) * sizeof(double));
    hipMalloc(&d_B, batch_size * in_size * (grid_size + spline_order) * sizeof(double));
    hipMalloc(&d_coefs, in_size * (grid_size + spline_order) * out_size * sizeof(double));
    hipMalloc(&d_knots, (n + spline_order + 1) * sizeof(double));

    hipMemcpy(d_x, x, batch_size * in_size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, batch_size * in_size * out_size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_grid, grid, in_size * (grid_size + 2 * spline_order + 1) * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_knots, h_knots, (n + spline_order + 1) * sizeof(double), hipMemcpyHostToDevice);

    dim3 blockDim(16, 16);
    dim3 gridDim((in_size + blockDim.x - 1) / blockDim.x, (batch_size + blockDim.y - 1) / blockDim.y);

    calcSplineValuesKernel<<<gridDim, blockDim>>>(d_x, d_grid, grid_size, spline_order, d_B, batch_size, in_size, d_knots);

    dim3 blockDimCoefs(256);
    dim3 gridDimCoefs((in_size + blockDimCoefs.x - 1) / blockDimCoefs.x);

    fitCoefficientsKernel<<<gridDimCoefs, blockDimCoefs>>>(d_B, d_y, d_coefs, batch_size, in_size, grid_size, spline_order, out_size, l2_reg);

    double* h_coefs = (double*)calloc(in_size * (grid_size + spline_order - 1) * out_size, sizeof(double));

    hipMemcpy(h_coefs, d_coefs, in_size * (grid_size + spline_order - 1) * out_size * sizeof(double), hipMemcpyDeviceToHost);

    for (unsigned int i = 0; i < out_size; i++) {
        for (unsigned int j = 0; j < in_size; j++) {
            for (unsigned int k = 0; k < grid_size + spline_order - 1; k++) {
                layer->activation_functions[i][j][k] = h_coefs[i * in_size * (grid_size + spline_order - 1) + j * (grid_size + spline_order - 1) + k];
            }
        }
    }

    free(h_coefs);
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_grid);
    hipFree(d_B);
    hipFree(d_coefs);
    hipFree(d_knots);

    free(h_knots);
}